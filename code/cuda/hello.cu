// 2025-08-04 knowpd

// nvcc: NVidia Cuda Compliler

// How to complie:
//  nvcc hello.cu -o a.out


#include <hip/hip_runtime.h>
#include <iostream>

// Kernel function to run on GPU
__global__ void hello_from_gpu() {
    printf("Hello from GPU thread %d of %d\n", threadIdx.x, blockDim.x);
}

int main() {
    std::cout << "Launching kernel from CPU..." << std::endl;

    hello_from_gpu<<<2, 5>>>();       // <<<numBlocks, threadsPerBlock>>>
    hipDeviceSynchronize();          // Wait for GPU to finish
    return 0;
}

// OUTPUT:
//   $ ./a.out
//   Launching kernel from CPU...
//   Hello from GPU thread 0 of 5
//   Hello from GPU thread 1 of 5
//   Hello from GPU thread 2 of 5
//   Hello from GPU thread 3 of 5
//   Hello from GPU thread 4 of 5
//   Hello from GPU thread 0 of 5
//   Hello from GPU thread 1 of 5
//   Hello from GPU thread 2 of 5
//   Hello from GPU thread 3 of 5
//   Hello from GPU thread 4 of 5

// NOTE:
//   - A total of 10 threads are launched in parallel.
//   - This appears sequential, but it’s actually due to:
//     1. printf() being buffered or serialized — to avoid garbled console output, 
//        CUDA internally serializes printf output across threads.
//     2. Execution order is non-deterministic, but formatted output is printed in order to stdout.
//     3. GPU threads run in parallel, but printf does not reflect true execution timing.

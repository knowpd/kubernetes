// 2025-08-04 knowpd

// How to compile:
//   nvcc device_query.cu -o a.out


#include <hip/hip_runtime.h>
#include <iostream>
int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    std::cout << "CUDA-capable device count: " << deviceCount << std::endl;
    return 0;
}

// OUTPUT:
//   $ ./a.out
//   CUDA-capable device count: 1
